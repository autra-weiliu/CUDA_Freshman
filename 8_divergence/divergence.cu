#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel_1(float* values, const int len) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len) {
		if (index % 2 == 0) {
			values[index] = 1;
		} else {
			values[index] = 2;
		}
	}
}

__global__ void kernel_2(float* values, const int len) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int wrap_index = threadIdx.x / warpSize;
	if (index < len) {
		if (wrap_index % 2 == 0) {
			values[index] = 1;;
		} else {
			values[index] = 2;
		}
	}
}

__global__ void warmup_kernel(float* values, const int len) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len) {
		if (index % 2 == 0) {
			values[index] = 3;
		} else {
			values[index] = 4;
		}
	}
}

int main(int argc, char **argv)
{
	size_t default_device = 0;
	hipSetDevice(default_device);
	
	// gpu data
	float* a_d;
	float* b_d;
	size_t len = 100000000;
	size_t bytes_len = len * sizeof(float);

	// malloc data
	hipMalloc((float**) &a_d, bytes_len);
	hipMalloc((float**) &b_d, bytes_len);

	// try different kernel
	dim3 block_dim(512);
	dim3 grid_dim((len + block_dim.x - 1) / block_dim.x);

	// warmup first
	warmup_kernel<<<grid_dim, block_dim>>>(a_d, len);
	hipDeviceSynchronize();

	// execute kernel1
	float gpu_time;
	hipEvent_t start_1, stop_1, start_2, stop_2;
  	hipEventCreate(&start_1);
  	hipEventCreate(&stop_1);
  	hipEventRecord(start_1, 0);

	kernel_1<<<grid_dim, block_dim>>>(a_d, len);
	hipDeviceSynchronize();

	hipEventRecord(stop_1, 0);
	hipEventSynchronize(stop_1);
	hipEventElapsedTime(&gpu_time, start_1, stop_1);
	hipEventDestroy(start_1);
	hipEventDestroy(stop_1);

	printf("gpu time1: %fms \n", gpu_time);

	// execute kernel2
	hipEventCreate(&start_2);
  	hipEventCreate(&stop_2);
  	hipEventRecord(start_2, 0);

	kernel_2<<<grid_dim, block_dim>>>(a_d, len);
	hipDeviceSynchronize();

	hipEventRecord(stop_2, 0);
	hipEventSynchronize(stop_2);
	hipEventElapsedTime(&gpu_time, start_2, stop_2);
	hipEventDestroy(start_2);
	hipEventDestroy(stop_2);

	printf("gpu time2: %fms \n", gpu_time);

	return EXIT_SUCCESS;
}
