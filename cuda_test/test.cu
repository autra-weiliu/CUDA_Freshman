#include <iostream>
#include <hip/hip_runtime.h>


int main(){
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量" << devProp.multiProcessorCount << std::endl;
}
